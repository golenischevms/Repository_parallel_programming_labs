#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel для сложения векторов
__global__ void vectorAddGPU(const float *a, const float *b, float *c, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    size_t N = 1 << 20; // Размер вектора (например, 1 миллион элементов)
    size_t bytes = N * sizeof(float);

    // Выделение памяти на хосте
    float *h_a = (float *)malloc(bytes);
    float *h_b = (float *)malloc(bytes);
    float *h_c = (float *)malloc(bytes);

    // Инициализация данных
    for (size_t i = 0; i < N; ++i) {
        h_a[i] = 1.0f;  // Первый вектор
        h_b[i] = 2.0f;  // Второй вектор
    }

    // Выделение памяти на устройстве
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Копирование данных с хоста на устройство
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Настройка сетки и блоков
    int threads = 256;                       // Количество нитей в блоке
    int blocks = (N + threads - 1) / threads; // Количество блоков

    // Измерение времени выполнения на GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAddGPU<<<blocks, threads>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    // Ожидание завершения всех вычислений на GPU
    hipDeviceSynchronize();

    // Копирование результата обратно на хост
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Размер вектора: %zu\n", N);
    printf("Время выполнения на GPU: %f ms\n", milliseconds);

    // Освобождение памяти
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}


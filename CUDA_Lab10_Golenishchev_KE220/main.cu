#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel для сложения векторов
__global__ void vectorAddGPU(const float *a, const float *b, float *c, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

void measurePerformance(size_t N) {
    size_t bytes = N * sizeof(float);

    // Выделение памяти на хосте
    float *h_a = (float *)malloc(bytes);
    float *h_b = (float *)malloc(bytes);
    float *h_c = (float *)malloc(bytes);

    // Инициализация данных
    for (size_t i = 0; i < N; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Выделение памяти на устройстве
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Копирование данных с хоста на устройство
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Настройка сетки и блоков
    int threads = 256;  // Количество потоков в блоке
    int blocks = (N + threads - 1) / threads; // Количество блоков

    // Измерение времени выполнения на GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAddGPU<<<blocks, threads>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    // Ожидание завершения всех вычислений на GPU
    hipDeviceSynchronize();

    // Копирование результата обратно на хост
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Вывод параметров
    printf("Размер вектора: %zu\n", N);
    printf("Количество блоков: %d\n", blocks);
    printf("Количество потоков в блоке: %d\n", threads);
    printf("Время выполнения на GPU: %f ms\n\n", milliseconds);

    // Освобождение памяти
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    // Тестирование для различных размеров векторов
    size_t sizes[] = {1 << 10, 1 << 15, 1 << 20, 1 << 25};
    int numSizes = sizeof(sizes) / sizeof(sizes[0]);

    for (int i = 0; i < numSizes; ++i) {
        measurePerformance(sizes[i]);
    }

    return 0;
}


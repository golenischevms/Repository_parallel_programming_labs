#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// Оригинальная функция для вычисления числа π на CPU
double CalcPi(const int n) {
    double pi = 0;
    const double coef = 1.0 / n;

    for (int i = 0; i < n; ++i) {
        const double xi = (i + 0.5) * coef;
        pi += 4.0 / (1.0 + xi * xi);
    }
    return pi * coef;
}

// CUDA ядро для параллельного вычисления суммы
__global__ void calcPiKernel(const int n, double* partialSums) {
    extern __shared__ double sharedSums[]; // Shared memory для частичных сумм

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    const double coef = 1.0 / n;
    sharedSums[tid] = 0.0;

    if (idx < n) {
        const double xi = (idx + 0.5) * coef;
        sharedSums[tid] = 4.0 / (1.0 + xi * xi);
    }

    __syncthreads();

    // Редукция внутри блока
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedSums[tid] += sharedSums[tid + stride];
        }
        __syncthreads();
    }

    // Запись результата блока в глобальную память
    if (tid == 0) {
        partialSums[blockIdx.x] = sharedSums[0];
    }
}

// Функция для вычисления числа π на GPU
double CalcPiGPU(const int n) {
    const int blockSize = 256; // Количество потоков в блоке
    const int numBlocks = (n + blockSize - 1) / blockSize; // Количество блоков

    // Выделение памяти на устройстве
    double* d_partialSums;
    hipMalloc(&d_partialSums, numBlocks * sizeof(double));

    // Запуск CUDA ядра
    calcPiKernel<<<numBlocks, blockSize, blockSize * sizeof(double)>>>(n, d_partialSums);

    // Копирование частичных сумм на хост
    double* h_partialSums = new double[numBlocks];
    hipMemcpy(h_partialSums, d_partialSums, numBlocks * sizeof(double), hipMemcpyDeviceToHost);

    // Редукция на хосте
    double pi = 0.0;
    for (int i = 0; i < numBlocks; ++i) {
        pi += h_partialSums[i];
    }

    // Освобождение памяти
    delete[] h_partialSums;
    hipFree(d_partialSums);

    return pi * (1.0 / n);
}

int main() {
    const int n = 100000000; // Количество интервалов для вычисления π

    // CPU вычисление
    auto startCPU = std::chrono::high_resolution_clock::now();
    double piCPU = CalcPi(n);
    auto endCPU = std::chrono::high_resolution_clock::now();
    double cpuTime = std::chrono::duration<double>(endCPU - startCPU).count();

    std::cout << "Computed pi (CPU): " << piCPU << ", Time: " << cpuTime << " seconds\n";

    // GPU вычисление
    float gpuTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    double piGPU = CalcPiGPU(n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpuTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Computed pi (GPU): " << piGPU << ", Time: " << gpuTime / 1000.0 << " seconds\n";

    return 0;
}
